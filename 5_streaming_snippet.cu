#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void device_kernel_add() {
  printf("Does add on device");
}

__global__ void device_kernel_mul() {
  printf("Does mul on device");
}

void host_func_add() {
  printf("Does add on host");
}

void host_func_mul() {
  printf("Does mul on host");
}

int main()
{
  //timer.start();

  uint64_t * host_ptr, * device_ptr;

  const uint64_t size = 64;

  int grid = 1;
  int block = 1;

  hipStream_t s_1, s_2;
  hipStreamCreate(&s_1);
  hipStreamCreate(&s_2);

  hipMemcpyAsync(host_ptr, device_ptr, size, hipMemcpyDeviceToHost, s_2);
  hipMemcpyAsync(host_ptr, device_ptr, size, hipMemcpyDeviceToHost, s_1);

  device_kernel_add<<<grid, block, 0, s_1>>>();
  device_kernel_add<<<grid, block, 0, s_2>>>();
  device_kernel_mul<<<grid, block, 0, s_1>>>();
  device_kernel_mul<<<grid, block, 0, s_2>>>();

  hipMemcpyAsync(device_ptr, host_ptr, size, hipMemcpyHostToDevice, s_2);
  hipMemcpyAsync(device_ptr, host_ptr, size, hipMemcpyHostToDevice, s_1);

  hipStreamSynchronize(s_1);
  hipStreamSynchronize(s_2);

  hipStreamDestroy(s_1);
  hipStreamDestroy(s_2);

}

//custom created streams run concurrently (s_1 and s_2 run independently of each other), but tasks scheduled on each stream execute sequentially, respecting the order in which they were scheduled.
