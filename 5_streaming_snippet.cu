hipStream_t s_1, s_2;
hipStreamCreate(&s_1);
hipStreamCreate(&s_2);

hipMemcpyAsync(host_ptr, device_ptr, size, hipMemcpyDeviceToHost, s_2);
hipMemcpyAsync(host_ptr, device_ptr, size, hipMemcpyDeviceToHost, s_1);

device_kernel_add<<<grid, block, 0, s_1>>>();
device_kernel_add<<<grid, block, 0, s_2>>>();
device_kernel_mul<<<grid, block, 0, s_1>>>();
device_kernel_mul<<<grid, block, 0, s_2>>>();

hipMemcpyAsync(device_ptr, host_ptr, size, hipMemcpyHostToDevice, s_2);
hipMemcpyAsync(device_ptr, host_ptr, size, hipMemcpyHostToDevice, s_1);

hipStreamSynchronize(s_1);
hipStreamSynchronize(s_2);

hipStreamDestroy(s_1);
hipStreamDestroy(s_2);
