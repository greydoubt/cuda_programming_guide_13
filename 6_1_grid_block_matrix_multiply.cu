/*
To define either grids or blocks with two or 3 dimensions, use CUDA's dim3 type as such:
  dim3 threads_per_block(16, 16, 1);
  dim3 number_of_blocks(16, 16, 1);
  someKernel<<<number_of_blocks, threads_per_block>>>();

Given the example just above, the variables gridDim.x, gridDim.y, blockDim.x, and blockDim.y inside of someKernel, would all be equal to 16.

You will need to create an execution configuration whose arguments are both dim3 values with the x and y dimensions set to greater than 1.
Inside the body of the kernel, you will need to establish the running thread's unique index within the grid per usual, but you should establish two indices for the thread: one for the x axis of the grid, and one for the y axis of the grid.

*/

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N  64

__global__ void matrixMulGPU( int * a, int * b, int * c )
{
  int val = 0;

  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < N && col < N)
  {
    for ( int k = 0; k < N; ++k )
      val += a[row * N + k] * b[k * N + col];
    c[row * N + col] = val;
  }
}

void matrixMulCPU( int * a, int * b, int * c )
{
  int val = 0;

  for( int row = 0; row < N; ++row )
    for( int col = 0; col < N; ++col )
    {
      val = 0;
      for ( int k = 0; k < N; ++k )
        val += a[row * N + k] * b[k * N + col];
      c[row * N + col] = val;
    }
}

int main()
{
  int *a, *b, *c_cpu, *c_gpu;

  int size = N * N * sizeof (int); // Number of bytes of an N x N matrix

  // Allocate memory
  hipMallocManaged (&a, size);
  hipMallocManaged (&b, size);
  hipMallocManaged (&c_cpu, size);
  hipMallocManaged (&c_gpu, size);

  // Initialize memory
  for( int row = 0; row < N; ++row )
    for( int col = 0; col < N; ++col )
    {
      a[row*N + col] = row;
      b[row*N + col] = col+2;
      c_cpu[row*N + col] = 0;
      c_gpu[row*N + col] = 0;
    }

  dim3 threads_per_block (16, 16, 1); // A 16 x 16 block threads
  dim3 number_of_blocks ((N / threads_per_block.x) + 1, (N / threads_per_block.y) + 1, 1);

  matrixMulGPU <<< number_of_blocks, threads_per_block >>> ( a, b, c_gpu );

  hipDeviceSynchronize(); // Wait for the GPU to finish before proceeding

  // Call the CPU version to check our work
  matrixMulCPU( a, b, c_cpu );

  // Compare the two answers to make sure they are equal
  bool error = false;
  for( int row = 0; row < N && !error; ++row )
    for( int col = 0; col < N && !error; ++col )
      if (c_cpu[row * N + col] != c_gpu[row * N + col])
      {
        printf("FOUND ERROR at c[%d][%d]\n", row, col);
        error = true;
        break;
      }
  if (!error)
    printf("Success!\n");

  // Free all our allocated memory
  hipFree(a); hipFree(b);
  hipFree( c_cpu ); hipFree( c_gpu );
}
